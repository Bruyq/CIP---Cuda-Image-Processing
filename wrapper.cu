#include "hip/hip_runtime.h"
#include "wrapper.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


// Applies mean filter to an image (Wrapper function for corresponding kernel)
// Parameters :
// - "radius" for local averaging kernel radius
// - "dst" image is destination image, "src" image is source image
__host__ void meanFilter(Image& dst, Image& src, int radius)
{
    // Verify that sizes are the same
    if (dst.getWidth() != src.getWidth() || dst.getHeight() != src.getHeight() || dst.getChannels() != src.getChannels() || radius > src.getWidth() / 2 || radius > src.getHeight() / 2)
    {
        std::cout << "Input and output images don't have the same dimensions or radius is too high" << std::endl;
        return;
    }
    else
    {
        // Initialisation
        size_t baseSize = src.getSize() * sizeof(unsigned char);
        int paddedWidth = src.getWidth() + 2 * radius;
        int paddedHeight = src.getHeight() + 2 * radius;
        size_t paddedSize = paddedWidth * paddedHeight * src.getChannels() * sizeof(unsigned char);
        unsigned char* d_padded1, * d_padded2, * d_src;

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_padded1, paddedSize));
        checkCudaErrors(hipMalloc((void**)&d_padded2, paddedSize));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), baseSize, hipMemcpyHostToDevice));

        // Extand base array to have the correct result on borders
        replicate(d_padded1, d_src, src.getWidth(), src.getHeight(), radius, src.getChannels());

        // Run device function
        dim3 block_dim(32, 32);
        dim3 grid_dim((paddedWidth * src.getChannels() + block_dim.x - 1) / block_dim.x, (paddedHeight + block_dim.y - 1) / block_dim.y);
        clock_t timer = clock();
        meanFilterKernel << <grid_dim, block_dim >> > (d_padded2, d_padded1, paddedWidth, paddedHeight, radius, src.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        std::cout << "Duration of meanKernel : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl;


        // Crop the result to get relevant data
        crop(d_src, d_padded2, radius, radius, src.getWidth(), src.getHeight(), paddedWidth, src.getChannels());

        // Retrieve result to host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_src, baseSize, hipMemcpyDeviceToHost));

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_padded1));
        checkCudaErrors(hipFree(d_padded2));
    }
}


// Applies laplacian filter to an image (Wrapper function for corresponding kernel)
// Parameters :
// - "dst" image is destination image, "src" image is source image
__host__ void laplacianFilter(Image& dst, Image& src)
{
    // Verify that sizes are the same
    if (dst.getWidth() != src.getWidth() || dst.getHeight() != src.getHeight() || dst.getChannels() != src.getChannels())
    {
        std::cout << "Input and output images don't have the same dimensions or radius is too high" << std::endl;
        return;
    }
    else
    {
        // Initialisation
        size_t baseSize = src.getSize() * sizeof(unsigned char);
        int paddedWidth = src.getWidth() + 2;
        int paddedHeight = src.getHeight() + 2;
        size_t paddedSize = paddedWidth * paddedHeight * src.getChannels() * sizeof(unsigned char);
        unsigned char* d_padded1, * d_padded2, * d_src;

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_padded1, paddedSize));
        checkCudaErrors(hipMalloc((void**)&d_padded2, paddedSize));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), baseSize, hipMemcpyHostToDevice));

        // Extand base array to have the correct result on borders
        replicate(d_padded1, d_src, src.getWidth(), src.getHeight(), 1, src.getChannels());

        // Run device function
        dim3 block_dim(32, 32);
        dim3 grid_dim((paddedWidth * src.getChannels() + block_dim.x - 1) / block_dim.x, (paddedHeight + block_dim.y - 1) / block_dim.y);
        clock_t timer = clock();
        laplacianFilterKernel << <grid_dim, block_dim >> > (d_padded2, d_padded1, paddedWidth, paddedHeight, src.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        std::cout << "Duration of laplacianKernel : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl;


        // Crop the result to get relevant data
        crop(d_src, d_padded2, 1, 1, src.getWidth(), src.getHeight(), paddedWidth, src.getChannels());

        // Retrieve result to host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_src, baseSize, hipMemcpyDeviceToHost));

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_padded1));
        checkCudaErrors(hipFree(d_padded2));
    }
}


// Generates an image displaying color gradients (Wrapper function for corresponding kernel)
// Parameters :
// - "img" is the result of this function
__host__ void generateRGB(Image& img)
{
    // Initialisation
    size_t arraySize = img.getSize() * sizeof(unsigned char);
    unsigned char* d_dest;
    dim3 block_dim(3, 1);
    dim3 grid_dim((img.getWidth() * img.getChannels() + block_dim.x - 1) / block_dim.x, (img.getHeight() + block_dim.y - 1) / block_dim.y);

    // Memory allocation on device
    checkCudaErrors(hipMalloc((void**)&d_dest, arraySize));

    // Run device function
    generateRGBKernel << <grid_dim, block_dim >> > (d_dest, img.getWidth(), img.getHeight());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(img.getData(), d_dest, arraySize, hipMemcpyDeviceToHost));

    // Free memory
    checkCudaErrors(hipFree(d_dest));
}


// Pads image with "replicate" option  (Wrapper function for corresponding kernel)
// Parameters :
// - "radius" for padding length
// - "dst" is for destination image , "src" is for source image
__host__ void replicate(Image& dst, Image& src, int radius)
{
    // Verify radius is below (img1_dims/2 + 1) and that img1 dims are superior to img2 dims
    if (radius > src.getWidth() - 1 || radius > src.getHeight() - 1 || src.getWidth() + 2 * radius != dst.getWidth() || src.getHeight() + 2 * radius != dst.getHeight())
    {
        std::cout <<
            "Case not handled : either the radius is too high for this image, either the dimensions of the images are invalid"
            << std::endl;
    }
    else
    {
        // Initialisation
        unsigned char* d_dest, * d_src;

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, src.getSize() * sizeof(unsigned char)));
        checkCudaErrors(hipMalloc((void**)&d_dest, dst.getSize() * sizeof(unsigned char)));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), src.getSize() * sizeof(unsigned char), hipMemcpyHostToDevice));

        // Call device function
        replicate(d_dest, d_src, src.getWidth(), src.getHeight(), radius, src.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        // Retrieve in host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_dest, dst.getSize() * sizeof(unsigned char), hipMemcpyDeviceToHost));

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_dest));
    }
}


// Pads image data with "replicate" option  (Wrapper function for corresponding kernel)
// Parameters :
// - "width", "height", "channels" for image dimensions
// - "radius" for padding length
// - "d_dest" device data is for destination data , "d_src" device data is for source data
__host__ void replicate(unsigned char* d_dest, unsigned char* d_src, int width, int height, int radius, int channels)
{
    // Initialisation
    clock_t timer;
    dim3 block_dim(32, 32);
    dim3 grid_dim(((width + 2 * radius) * channels + block_dim.x - 1) / block_dim.x, (height + 2 * radius + block_dim.y - 1) / block_dim.y);

    // Run device function
    timer = clock();
    replicateKernel << <grid_dim, block_dim >> > (d_dest, d_src, width, height, radius, channels);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    std::cout << "Duration of replicateKernel : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl; // 512x512 -> 1024x1024 in 5ms
}


// Crops an image (Wrapper function for corresponding kernel)
// Parameters :
// - "width", "height" are for destination image dimensions
// - "(posX, posY)" is upper-left starting point for cropping in source image
// - "dst" is for destination image , "src" is for source image
__host__ void crop(Image& dst, Image& src, int posX, int posY, int width, int height)
{
    // Verify size requirements
    if (src.getWidth() <= width + posX || src.getHeight() <= height + posY || dst.getWidth() > width || dst.getHeight() > height)
    {
        std::cout <<
            "Can't crop : Invalid dimensions"
            << std::endl;
    }
    else
    {
        // Initialisation
        unsigned char* d_dest, * d_src;

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, src.getSize() * sizeof(unsigned char)));
        checkCudaErrors(hipMalloc((void**)&d_dest, dst.getSize() * sizeof(unsigned char)));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), src.getSize() * sizeof(unsigned char), hipMemcpyHostToDevice));

        // Call device function
        crop(d_dest, d_src, posX, posY, dst.getWidth(), dst.getHeight(), src.getWidth(), dst.getChannels());

        // Retrieve in host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_dest, dst.getSize() * sizeof(unsigned char), hipMemcpyDeviceToHost));

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_dest));
    }
}


// Crops an image (Wrapper function for corresponding kernel)
// Parameters :
// - "width", "height", "channels" are for destination image data dimensions
// - "widthInit" is for source image width
// - "(posX, posY)" is upper-left starting point for cropping in source image data
// - "d_dest" device data is for destination data , "d_src" device data is for source data
__host__ void crop(unsigned char* d_dest, unsigned char* d_src, int posX, int posY, int width, int height, int widthInit, int channels)
{
    // Initialisation
    clock_t timer;
    dim3 block_dim(32, 32);
    dim3 grid_dim((width * channels + block_dim.x - 1) / block_dim.x, (height + block_dim.y - 1) / block_dim.y);

    // Run device function
    timer = clock();
    cropKernel << <grid_dim, block_dim >> > (d_dest, d_src, posX, posY, width, height, widthInit, channels); // very fast - depends on the case ?
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    std::cout << "Duration of cropKernel : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl;
}


// Applies guided filter smoothing technique
// Parameters :
// - "radius" for kernel radius
// - "dst" is for destination image , "src" is for source image
__host__ void guidedFilterSmoothing(Image& dst, Image& src, int radius)
{
    // Verify that sizes are the same
    if (dst.getWidth() != src.getWidth() || dst.getHeight() != src.getHeight() || dst.getChannels() != src.getChannels() || 2 * radius > src.getWidth() / 2 || 2 * radius > src.getHeight() / 2)
    {
        std::cout << "Input and output images don't have the same dimensions or radius is too high" << std::endl;
        return;
    }
    else
    {
        // Initialisation
        size_t baseSize = src.getSize() * sizeof(unsigned char);
        int paddedWidth = src.getWidth() + 4 * radius;
        int paddedHeight = src.getHeight() + 4 * radius;
        size_t paddedSizeUC = paddedWidth * paddedHeight * src.getChannels() * sizeof(unsigned char);
        size_t paddedSizeF = paddedWidth * paddedHeight * src.getChannels() * sizeof(float);
        unsigned char* d_padded1, * d_padded2, * d_src;
        float* d_ak, * d_bk;
        clock_t timer = clock();

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_padded1, paddedSizeUC));
        checkCudaErrors(hipMalloc((void**)&d_padded2, paddedSizeUC));
        checkCudaErrors(hipMalloc((void**)&d_ak, paddedSizeF));
        checkCudaErrors(hipMalloc((void**)&d_bk, paddedSizeF));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), baseSize, hipMemcpyHostToDevice));

        // Extand base array to have the correct result on borders
        replicate(d_padded1, d_src, src.getWidth(), src.getHeight(), 2 * radius, src.getChannels());

        // Run device function
        dim3 block_dim(32, 32);
        dim3 grid_dim((paddedWidth * src.getChannels() + block_dim.x - 1) / block_dim.x, (paddedHeight + block_dim.y - 1) / block_dim.y);
        guidedFirstKernel << <grid_dim, block_dim >> > (d_ak, d_bk, d_padded1, paddedWidth, paddedHeight, radius, src.getChannels());
        checkCudaErrors(hipGetLastError());
        guidedSecondKernel << <grid_dim, block_dim >> > (d_padded2, d_ak, d_bk, d_padded1, paddedWidth, paddedHeight, radius, src.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        // Crop the result to get relevant data
        crop(d_src, d_padded2, 2 * radius, 2 * radius, src.getWidth(), src.getHeight(), paddedWidth, src.getChannels());

        // Retrieve result to host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_src, baseSize, hipMemcpyDeviceToHost));
        std::cout << "Duration of guidedFilterSmoothing : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl;

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_padded1));
        checkCudaErrors(hipFree(d_padded2));
        checkCudaErrors(hipFree(d_ak));
        checkCudaErrors(hipFree(d_bk));
    }
}


// Applies guided filter detail enhancement technique
// Parameters :
// - "radius" for kernel radius
// - "value" is for enhancement factor value
// - "dst" is for destination image , "src" is for source image
__host__ void guidedFilterEnhancement(Image& dst, Image& src, int radius, float value)
{
    // Verify that sizes are the same
    if (dst.getWidth() != src.getWidth() || dst.getHeight() != src.getHeight() || dst.getChannels() != src.getChannels() || 2 * radius > src.getWidth() / 2 || 2 * radius > src.getHeight() / 2)
    {
        std::cout << "Input and output images don't have the same dimensions or radius is too high" << std::endl;
        return;
    }
    else
    {
        // Initialisation
        size_t baseSize = src.getSize() * sizeof(unsigned char);
        int paddedWidth = src.getWidth() + 4 * radius;
        int paddedHeight = src.getHeight() + 4 * radius;
        size_t paddedSizeUC = paddedWidth * paddedHeight * src.getChannels() * sizeof(unsigned char);
        size_t paddedSizeF = paddedWidth * paddedHeight * src.getChannels() * sizeof(float);
        unsigned char* d_padded1, * d_padded2, * d_src, * d_src2, * d_dst;
        float* d_ak, * d_bk;
        clock_t timer = clock();

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_src2, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_dst, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_padded1, paddedSizeUC));
        checkCudaErrors(hipMalloc((void**)&d_padded2, paddedSizeUC));
        checkCudaErrors(hipMalloc((void**)&d_ak, paddedSizeF));
        checkCudaErrors(hipMalloc((void**)&d_bk, paddedSizeF));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), baseSize, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_src2, src.getData(), baseSize, hipMemcpyHostToDevice));

        // Extand base array to have the correct result on borders
        replicate(d_padded1, d_src, src.getWidth(), src.getHeight(), 2 * radius, src.getChannels());

        // Run device function
        dim3 block_dim(32, 32);
        dim3 grid_dim((paddedWidth * src.getChannels() + block_dim.x - 1) / block_dim.x, (paddedHeight + block_dim.y - 1) / block_dim.y);
        guidedFirstKernel << <grid_dim, block_dim >> > (d_ak, d_bk, d_padded1, paddedWidth, paddedHeight, radius, src.getChannels());
        checkCudaErrors(hipGetLastError());
        guidedSecondKernel << <grid_dim, block_dim >> > (d_padded2, d_ak, d_bk, d_padded1, paddedWidth, paddedHeight, radius, src.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        // Crop the result to get relevant data
        crop(d_src, d_padded2, 2 * radius, 2 * radius, src.getWidth(), src.getHeight(), paddedWidth, src.getChannels());

        grid_dim.x = (src.getWidth() * src.getChannels() + block_dim.x - 1) / block_dim.x;
        grid_dim.y = (src.getHeight() + block_dim.y - 1) / block_dim.y;
        unsharpMaskingKernel << <grid_dim, block_dim >> > (d_dst, d_src2, d_src, dst.getWidth(), dst.getHeight(), value, dst.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        // Retrieve result to host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_dst, baseSize, hipMemcpyDeviceToHost));
        std::cout << "Duration of guidedFilterEnhancement : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl;

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_src2));
        checkCudaErrors(hipFree(d_dst));
        checkCudaErrors(hipFree(d_padded1));
        checkCudaErrors(hipFree(d_padded2));
        checkCudaErrors(hipFree(d_ak));
        checkCudaErrors(hipFree(d_bk));
    }
}


// Binarizes an image based on the value of a specified channel (Wrapper function for corresponding kernel)
// Parameters :
// - "target_channel" is an integer that selects the channel to work on : 0 for red channel, 2 for blue, any other value for green
// - "threshold" selects the threshold value. Values higher or equal will be white (255), other values will be black (0). "threshold" should be in [0, 255]
// - "dst" is for destination image , "src" is for source image
__host__ void binarize(Image& dst, Image& src, int target_channel, int threshold)
{
    // Verify that sizes are the same
    if (dst.getWidth() != src.getWidth() || dst.getHeight() != src.getHeight() || dst.getChannels() != src.getChannels())
    {
        std::cout << "Input and output images don't have the same dimensions" << std::endl;
        return;
    }
    else
    {
        // Initialization
        size_t baseSize = src.getSize() * sizeof(unsigned char);
        unsigned char* d_dst, * d_src;

        // Memory allocation on device
        checkCudaErrors(hipMalloc((void**)&d_src, baseSize));
        checkCudaErrors(hipMalloc((void**)&d_dst, baseSize));

        // Copy to device memory
        checkCudaErrors(hipMemcpy(d_src, src.getData(), baseSize, hipMemcpyHostToDevice));

        // Run device function
        dim3 block_dim(32, 32);
        dim3 grid_dim((src.getWidth() + block_dim.x - 1) / block_dim.x, (src.getHeight() + block_dim.y - 1) / block_dim.y);
        clock_t timer = clock();
        binarizeKernel << <grid_dim, block_dim >> > (d_dst, d_src, src.getWidth(), src.getHeight(), target_channel, threshold, src.getChannels());
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        std::cout << "Duration of binarizeKernel : " << (float)(clock() - timer) / CLOCKS_PER_SEC << " seconds" << std::endl;

        // Retrieve result to host memory
        checkCudaErrors(hipMemcpy(dst.getData(), d_dst, baseSize, hipMemcpyDeviceToHost));

        // Free memory
        checkCudaErrors(hipFree(d_src));
        checkCudaErrors(hipFree(d_dst));
    }
}