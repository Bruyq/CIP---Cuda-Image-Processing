#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

// Applies mean filter to the data
// Parameters : 
// - "width", "height", "channels" for image dimensions
// - "radius" for mean filter radius
// - "dest" stands for destination data and "src" for source data
__global__ void meanFilterKernel(unsigned char* dest, const unsigned char* src, int width, int height, int radius, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width * channels && y < height)
    {
        // if RGB image : One line has RGB values for each pixels, so width is in fact width * 3
        int index = (y * width * channels + x);
        float sum = 0.0f;

        for (int i = -radius; i <= radius; i++) {
            for (int j = -radius; j <= radius; j++) 
            {
                // Following x axis we need to multiply the offset by the number of channels to avoid comparison between two distinct channels
                int px = x + i * channels;
                int py = y + j;

                if (px >= 0 && px < width * channels && py >= 0 && py < height)
                {
                    sum += (float)src[py * width * channels + px];
                }
            }
        }
        dest[index] = (unsigned char)(sum / ((2 * radius + 1) * (2 * radius + 1)));
    }
}


// Applies unsharp masking to the data (performs detail enhancement)
// Parameters : 
// - "width", "height", "channels" for image dimensions
// - "factor" for detail amplification factor
// - "dest" stands for destination data, "src" for source data and "smoothed" is the smoothed version of src data
__global__ void unsharpMaskingKernel(unsigned char* dest, const unsigned char* src, const unsigned char* smoothed, int width, int height, float factor, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width * channels && y < height)
    {
        // if RGB image : One line has RGB values for each pixels, so width is in fact width * 3
        int index = (y * width * channels + x);
        float val = src[index];
        float res = val + factor * (val - smoothed[index]);
        
        dest[index] = (unsigned char)(res > 255 ? 255 : res);
    }
}


// Applies laplacian filter to the data (useful for edge detection)
// Parameters : 
// - "width", "height", "channels" for image dimensions
// - "dest" stands for destination data and "src" for source data
__global__ void laplacianFilterKernel(unsigned char* dest, const unsigned char* src, int width, int height, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float kernel[3][3] = { {0, -1, 0}, {-1, 4, -1}, {0, -1, 0} };
    //float kernel[3][3] = { {-1, -1, -1}, {-1, 8, -1}, {-1, -1, -1} };

    if (x < width * channels && y < height)
    {
        // if RGB image : One line has RGB values for each pixels, so width is in fact width * 3
        int index = (y * width * channels + x);
        float sum = 0.0f;

        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++)
            {
                // Following x axis we need to multiply the offset by the number of channels to avoid comparison between two distinct channels
                int px = x + i * channels;
                int py = y + j;

                if (px >= 0 && px < width * channels && py >= 0 && py < height)
                {
                    sum += (float)src[py * width * channels + px] * kernel[i + 1][j + 1];
                }
            }
        }
        dest[index] = (unsigned char)(sum);
    }
}


// Generates test image data displaying color gradients
// Parameters : 
// - "width", "height", "channels" for image dimensions
// - "dest" stands for destination data and "src" for source data
__global__ void generateRGBKernel(unsigned char* dest, int width, int height)
{
    // Block dim must be (3, 1) so we can know the channel we are working on
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int factor = 3; // 3 RGB values
    if (x < width * factor && y < height)
    {
        // RGB image : One line has RGB values for each pixels, so width is in fact width * 3
        int index = (y * width * factor + x);
        if (threadIdx.x == 2)
        {
            // Blue channel is null
            dest[index] = (unsigned char)(0);
        }
        else
        {
            // Red channel increments following X direction, green channel increments following y direction
            dest[index] = (unsigned char)((threadIdx.x == 0) ? blockIdx.x : blockIdx.y);
        }
    }
}


// Pads image data with "replicate" option 
// Example : 
// - (10, 0, 23), (50, 60, 41), (23, 23, 23) | (23, 23, 23), (50, 60, 41), (10, 0, 23) where "|" is initial image border
// Parameters :
// - "width", "height", "channels" for image dimensions
// - "radius" is equal to the padding length on borders
// - "dest" stands for destination data and "src" for source data
__global__ void replicateKernel(unsigned char* dest, const unsigned char* src, int width, int height, int radius, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = (y * (width + 2 * radius) * channels + x);

    // If image as 3 channels, data will be like : RGBRGBRGB... and so on, so we adapt the algorithm for 3 cases
    if (y < radius && x < radius * channels)                                                                                    // Top-left corner
    {
        if ((x - 2) % 3 == 0 && channels == 3)
		{
			// When it is supposed to be blue
			dest[index] = src[radius * channels - x + (radius - y) * width * channels + 1];
		}
		else if (x % 3 == 0 && channels == 3)
		{
			// When it is supposed to be red
			dest[index] = src[radius * channels - x + (radius - y) * width * channels - 3];
		}
		else
		{
			// When it is supposed to be green (no modification on the behavior, same as if it was a 1 channel image)
			dest[index] = src[radius * channels - x + (radius - y) * width * channels - 1];
		}
    }
	else if (y < radius && x < (width + radius) * channels)                                                                     // Upper Border
	{
		dest[index] = src[(radius - y) * width * channels + x - radius * channels];
	}
	else if (y < radius && x < (width + 2 * radius) * channels)                                                                 // Top-right corner
	{
		if ((x - 2) % 3 == 0 && channels == 3)
		{
			// When it is supposed to be blue
			dest[index] = src[((radius - y + 1) * width + (width + radius)) * channels - x + 1];
		}
		else if (x % 3 == 0 && channels == 3)
		{
			// When it is supposed to be red
			dest[index] = src[((radius - y + 1) * width + (width + radius)) * channels - x - 3];
		}
		else
		{
			// When it is supposed to be green (no modification on the behavior, same as if it was a 1 channel image)
			dest[index] = src[((radius - y + 1) * width + (width + radius)) * channels - x - 1];
		}
	}
    else if (y < height + radius && x < radius * channels)                                                                      // Left border
    {
        if ((x - 2) % 3 == 0 && channels == 3)
        {
            // When it is supposed to be blue
            dest[index] = src[radius * channels - x + (y - radius) * width * channels + 1];
        }
        else if (x % 3 == 0 && channels == 3)
        {
            // When it is supposed to be red
            dest[index] = src[radius * channels - x + (y - radius) * width * channels - 3];
        }
        else
        {
            // When it is supposed to be green (no modification on the behavior, same as if it was a 1 channel image)
            dest[index] = src[radius * channels - x + (y - radius) * width * channels - 1];
        }
    }
    else if (y < height + radius && x < (width + radius) * channels)                                                            // Middle content
    {
        dest[index] = src[x - radius * channels + (y - radius) * width * channels];
    }
    else if (y < height + radius && x < (width + 2 * radius) * channels)                                                        // Right border
    {
        if ((x - 2) % 3 == 0 && channels == 3)
        {
            // When it is supposed to be blue
            dest[index] = src[((y - radius + 1) * width + (width + radius)) * channels - x + 1];
        }
        else if (x % 3 == 0 && channels == 3)
        {
            // When it is supposed to be red
            dest[index] = src[((y - radius + 1) * width + (width + radius)) * channels - x - 3];
        }
        else
        {
            // When it is supposed to be green (no modification on the behavior, same as if it was a 1 channel image)
            dest[index] = src[((y - radius + 1) * width + (width + radius)) * channels - x - 1];
        }
    }
    else if (y < 2 * radius + height && x < radius * channels)                                                                  // Bot-left corner
    {
		if ((x - 2) % 3 == 0 && channels == 3)
		{
			// When it is supposed to be blue
			dest[index] = src[(radius + 2 * height - y - 1) * width * channels + radius * channels - x + 1];
		}
		else if (x % 3 == 0 && channels == 3)
		{
			// When it is supposed to be red
			dest[index] = src[(radius + 2 * height - y - 1) * width * channels + radius * channels - x - 3];
		}
		else
		{
			// When it is supposed to be green (no modification on the behavior, same as if it was a 1 channel image)
			dest[index] = src[(radius + 2 * height - y - 1) * width * channels + radius * channels - x - 1];
		}
    }
    else if (y < 2 * radius + height && x < (width + radius) * channels)                                                        // Bottom border
    {
        dest[index] = src[(radius + 2 * height - y - 1) * width * channels + x - radius * channels];
    }
    else if (y < 2 * radius + height && x < (width + 2 * radius) * channels)                                                    // Bot-right corner
    {
        if ((x - 2) % 3 == 0 && channels == 3)
        {
            // When it is supposed to be blue
            dest[index] = src[((radius + 2 * height - y - 1) * width + (width + radius)) * channels - x + 1];
        }
        else if (x % 3 == 0 && channels == 3)
        {
            // When it is supposed to be red
            dest[index] = src[((radius + 2 * height - y - 1) * width + (width + radius)) * channels - x - 3];
        }
        else
        {
            // When it is supposed to be green (no modification on the behavior, same as if it was a 1 channel image)
            dest[index] = src[((radius + 2 * height - y - 1) * width + (width + radius)) * channels - x - 1];
        }
    }
}


// Crops image data
// Parameters :
// - "width", "height", "channels" for destination data dimensions
// - "widhtInit" for source data width and "(posX, posY)" is upper-left starting point for cropping in source image data
// - "dest" stands for destination data and "src" for source data
__global__ void cropKernel(unsigned char* dest, const unsigned char* src, int posX, int posY, int width, int height, int widthInit, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = (y * width * channels + x);

    if (x < width * channels && y < height)
    {
        dest[index] = src[((y + posY) * widthInit  + posX) * channels + x];
    }
}


// Applies the first step for guided filter smoothing technique
// Parameters :
// - "width", "height", "channels" for destination data dimensions
// - "radius" gives the kernel radius for local mean and local variance computation
// - "ak" and "bk" are kernel data outputs and "src" for source data
__global__ void guidedFirstKernel(float* ak, float* bk, const unsigned char* src, int width, int height, int radius, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width * channels && y < height)
    {
        // if RGB image : One line has RGB values for each pixels, so width is in fact width * 3
        int index = (y * width * channels + x);
        float mean = 0.0f;
        float w = (2 * radius + 1) * (2 * radius + 1);
        float var = 0.0f;
        float temp;

        for (int i = -radius; i <= radius; i++) 
        {
            for (int j = -radius; j <= radius; j++)
            {
                // Following x axis we need to multiply the offset by the number of channels to avoid comparison between two distinct channels
                int px = x + i * channels;
                int py = y + j;

                if (px >= 0 && px < width * channels && py >= 0 && py < height)
                {
                    temp = float(src[py * width * channels + px]) / 255;
                    mean += temp;
                    var += temp * temp;
                }
            }
        }
        mean = mean / w;
        var = var / w - mean * mean;
        ak[index] = ((var) / (var + 0.01f));
        bk[index] = ((1 - ak[index]) * mean);
    }
}


// Applies the second step for guided filter smoothing technique
// Parameters :
// - "width", "height", "channels" for destination data dimensions
// - "radius" gives the kernel radius for the local average computation
// - "dest" stands for destination data, "ak" and "bk" are data inputs and "src" stands for source data
__global__ void guidedSecondKernel(unsigned char* dest, const float* ak, const float* bk, const unsigned char* src, int width, int height, int radius, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width * channels && y < height)
    {
        // if RGB image : One line has RGB values for each pixels, so width is in fact width * 3
        int index = (y * width * channels + x);
        float a = 0.0f;
        float w = (2 * radius + 1) * (2 * radius + 1);
        float b = 0.0f;

        for (int i = -radius; i <= radius; i++) {
            for (int j = -radius; j <= radius; j++)
            {
                // Following x axis we need to multiply the offset by the numbre of channels to avoid comparison between two distinct channels
                int px = x + i * channels;
                int py = y + j;

                if (px >= 0 && px < width * channels && py >= 0 && py < height)
                {
                    a += ak[py * width * channels + px];
                    b += bk[py * width * channels + px];
                }
            }
        }
        a = a / w;
        b = b / w;
        dest[index] = (unsigned char)(a * float(src[index]) + b * 255);
    }
}


// Thresholds and binarize image data based on a specific channel.
// Parameters :
// - "width", "height", "channels" for destination data dimensions
// - "target_channel" is an integer that selects the channel to work on : 0 for red channel, 2 for blue, any other value for green
// - "threshold" selects the threshold value. Values higher or equal will be white (255), other values will be black (0). "treshold" should be in [0, 255]
// - "dest" stands for destination data and "src" for source data
__global__ void binarizeKernel(unsigned char* dest, unsigned char* src, int width, int height, int target_channel, int threshold, int channels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width * channels && y < height)
    {
        int index = (y * width + x) * channels;
 
        if (target_channel == 0 && channels == 3)                        // Thread working on red channel
        {
            if (src[index] >= threshold)
            {
                dest[index] = 255;
                dest[index + 1] = 255;
                dest[index + 2] = 255;
            }
            else
            {
                dest[index] = 0;
                dest[index + 1] = 0;
                dest[index + 2] = 0;
            }
        }
        else if (target_channel == 2 && channels == 3)             // Thread working on blue channel
        {
            if (src[index + 2] >= threshold)
            {
                dest[index] = 255;
                dest[index + 1] = 255;
                dest[index + 2] = 255;
            }
            else
            {
                dest[index] = 0;
                dest[index + 1] = 0;
                dest[index + 2] = 0;
            }
        }
        else if (channels == 3)                                    // Thread working on green channel
        {
            if (src[index + 1] >= threshold) 
            {
                dest[index] = 255;
                dest[index + 1] = 255;
                dest[index + 2] = 255;
            }
            else
            {
                dest[index] = 0;
                dest[index + 1] = 0;
                dest[index + 2] = 0;
            }
        }
        else                                                        // When working with 1 channel image
        {
            if (src[index] >= threshold)
            {
                dest[index] = 255;
            }
            else
            {
                dest[index] = 0;
            }
        }
    }
}

// TODO
__global__ void computeHistogramKernel(unsigned int* hist, unsigned char* src, int width, int height, int channels, int nbins)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width  && y < height)
    {
        int index = (y * width + x);
        if (channels == 1)
        {

        }
    }
}